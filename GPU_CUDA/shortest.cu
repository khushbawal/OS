// Elapsed Real Time for basis-5.txt:
// real    0m2.994s
// user    0m0.939s
// sys     0m1.916s
#include <stdio.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime.h>

// General function to report a failure and exit.
static void fail(char const *message)
{
  fprintf(stderr, "%s\n", message);
  exit(1);
}

// Print out a usage message, then exit.
static void usage()
{
  printf("usage: shortest [report]\n");
  exit(1);
}

// Maximum width of each basis vector.
#define DMAX 100

// Type that can be used to represent a basis vector.
typedef int BVec[DMAX];

// Actual length of each vector in the basis.
int width;

// List of all vectors defining the lattice.
BVec *basis;

// Number of vectors representing the basis.
int bCount = 0;

// Current capacity of the basis array
int bCap = 0;

// Read the input list of basis vectors.
void readBasis()
{
  // Determine the size of the basis
  if (scanf("%d", &width) != 1)
    fail("Can't read vector length");

  // Set up initial list and capacity.
  bCap = 5;
  basis = (BVec *)malloc(bCap * sizeof(BVec));

  // Read all the basis vectors from the input.
  int v;
  while (scanf("%d", &v) == 1)
  {
    // Grow the basis array if needed.
    if (bCount >= bCap)
    {
      bCap *= 2;
      basis = (BVec *)realloc(basis, bCap * sizeof(BVec));
    }

    // Fill in the next basis vector.
    basis[bCount][0] = v;
    for (int i = 1; i < width; i++)
      if (scanf("%d", &(basis[bCount][i])) != 1)
        fail("Can't read vector element");
    bCount += 1;
  }
}

__global__ void findShortest(BVec *devBasis, int *devResults, int bCount, int width, bool report)
{
  // Unique index for this worker.
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  if (i < bCount - 2 && j < width)
  {
    devResults[i] = 1000000000;
    // int shortestSum = 1000000000; // Initialize with a large value
    // Compute the locally shortest squared magnitude
    // Second Vector
    for (int k = i + 1; k < bCount - 1; k++)
    {
      // Third Vector
      for (int l = k + 1; l < bCount; l++)
      {
        int sum = 0;
        // Columns
        for (int m = 0; m < width; m++)
        {
          int sqrVal = (devBasis[i][j + m] + devBasis[k][j + m] + devBasis[l][j + m]);
          sum += (sqrVal * sqrVal);
        }
        // Check if local smaller
        if (sum < devResults[i])
        {
          devResults[i] = sum;
          // printf("i: %d shortestSum: %d\n", i, devResults[i]);
        }
        if (report == true)
        {
          printf("%d : | B[%d] + B[%d] + B[%d] | = %d\n", i, i, k, l, sum);
        }
      }
    }

    // Store the result in devResults
    // devResults[i] = shortestSum;
  }
  // for (int g = 0; g < sizeof(devResults); g++)
  // {
  //   printf("%d ", devResults[g]);
  // }
  // printf("\n");
}

int main(int argc, char *argv[])
{
  // If there's an argument, it better be "report"
  bool report = false;
  if (argc == 2)
  {
    if (strcmp(argv[1], "report") != 0)
      usage();
    report = true;
  }

  readBasis();

  // Allocate space on the device to hold a copy of the basis.
  BVec *devBasis = NULL;
  if (hipMalloc((void **)&devBasis, (bCount) * sizeof(BVec)) != hipSuccess)
  {
    fail("Failed to allocate space for lenght list on device");
  }
  // Copy the list over to the device.
  hipMemcpy(devBasis, basis, (bCount) * sizeof(BVec), hipMemcpyHostToDevice);
  // Allocate space on the device to hold the results.
  int *devResults = NULL;
  // long ncr = (bcount) * (bcount - 1) * (bcount - 2) / 6;
  if (hipMalloc((void **)&devResults, (bCount - 2) * sizeof(int)) != hipSuccess)
  {
    fail("Failed to allocate space for lenght list on device");
  }
  // Block and grid dimensions.
  int threadsPerBlock = 100;
  // Round up.
  int blocksPerGrid = (bCount + threadsPerBlock - 1) / threadsPerBlock;
  // Run our kernel on these block/grid dimensions
  findShortest<<<blocksPerGrid, threadsPerBlock>>>(devBasis, devResults, bCount, width, report);
  if (hipGetLastError() != hipSuccess)
    fail("Failure in CUDA kernel execution.");

  // Add code to copy the results list back to the host.
  int *results = (int *)malloc((bCount - 2) * sizeof(int));
  // printf("1: %d\n", sizeof(results));
  hipMemcpy(results, devResults, (bCount - 2) * sizeof(int), hipMemcpyDeviceToHost);
  // Find the minimum of all the results and report it.
  int small = results[0];
  // printf("2: %d , bCount: %d , sizeof(int): %d\n", sizeof(results), bCount, sizeof(int *));
  for (int i = 1; i < (bCount - 2); i++)
  {
    // printf("%d ", results[i]);
    if (results[i] < small && results[i] != 0)
    {
      small = results[i];
    }
  }
  printf("Shortest Vector: %.3f\n", sqrt(small));
  // Free memory on the device and the host.
  hipFree(devBasis);
  hipFree(devResults);
  free(results);
  hipDeviceReset();

  return 0;
}